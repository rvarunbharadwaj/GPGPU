﻿#include "GPUErrors.h"

bool HandleCUDAError(hipError_t t)
{
	if (t != hipSuccess)
	{
		cout << hipGetErrorString(hipGetLastError());
		return false;
	}
	return true;
}

bool GetCUDARunTimeError()
{
	hipError_t t = hipGetLastError();
	if (t != hipSuccess)
	{
		cout << hipGetErrorString(t) << endl;
		return false;
	}
	return true;
}

