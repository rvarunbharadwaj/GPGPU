#include "hip/hip_runtime.h"
#include "WarpFunc.h"

__global__ void CountOdds(int* g_Vect, int* g_Odds, const int Size)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;
	int localID = threadIdx.x;

	//Dynamic Shared Memory Allocation //Dynamic Shared memory can be only done in 1D
	extern __shared__ int count[]; //Vector of counters with size Block_sized

	//Using Thread Coarsening to have maximum active blocks per SM
	for (int i = idx; i < Size; i += totalThreads)
	{
		count[localID] += (g_Vect[i] % 2); //Size of the count should be block_size
	}
	//count[localID] += (g_Vect[idx] % 2); //Size of the count should be block_size
	__syncthreads();

	//Reduction phase: Summing up the block
	int step = 1;
	int otherIdx = localID | step; //Perform a bitwise OR
	while ((otherIdx < blockDim.x) && ((localID & step) == 0))
	{
		count[localID] += count[otherIdx];
		step <<= 1;
		otherIdx = localID | step;
		__syncthreads();
	}
	//Add to the global counts
	if (localID == 0)
	{
		atomicAdd(g_Odds, count[0]); //We'll use atomic add as many as blocks
	}
	
}

__host__ void CountOddsHelper(int* h, int oc_check, const int size)
{
	//Variable to store the GPU computed odd counts;
	int gpu_OddCount;
	//Device pointer for the input vector on the Global Memory
	int* d_vect{};
	//Device pointer for the odd count result from the kernel
	int* d_oc{};
	//Allocate Global Memory for the input vector and the odd count
	HandleCUDAError(hipMalloc((void**)&d_vect, VECTOR_SIZE_IN_BYTES));
	HandleCUDAError(hipMalloc((void**)&d_oc, sizeof(int) * 1)); //Just to store one value

	//Copy input data to the device memory
	HandleCUDAError(hipMemcpy(d_vect, h, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice));
	//Initialize the device odd count memory to zero
	HandleCUDAError(hipMemset(d_oc, 0, sizeof(int)*1));

	//Determining execution configuration for maximum active Blocks
	//Determine the number of SMs on the GPU
	hipDeviceProp_t prop;
	int device_id;
	HandleCUDAError(hipGetDevice(&device_id));
	HandleCUDAError(hipGetDeviceProperties(&prop, device_id));
	int SM = prop.multiProcessorCount;
	cout <<endl<< "Number of SMs: " << SM << endl;

	int blockSize = 256;
	int blockPerSM{}, gridSize{};
	//Determine the shared memory size for dynamic shared memory allocation
	int SharedMemSize = blockSize * sizeof(int); //1KB size cause 256 * 4 bytes
	//Call the execution configuration function hipOccupancyMaxActiveBlocksPerMultiprocessor API to determine blocks per SM
	hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockPerSM, (void*)CountOdds, blockSize, SharedMemSize);
	//Compute grid size
	gridSize = min((int)ceil(1.0 * SIZE / blockSize), blockPerSM * SM);
	//Display Execution Configuration
	cout << "Number of Threads per Block: " << blockSize << endl;
	cout << "Number of suggested blocks per SM for Maximum Active Blocks Per SM: " << blockPerSM << endl;
	cout << "Number of Blocks per Grid: " << gridSize << endl;

	CountOdds << <gridSize, blockSize, SharedMemSize >> > (d_vect, d_oc, size);
	hipDeviceSynchronize();
	HandleCUDAError(hipMemcpy(&gpu_OddCount, d_oc, sizeof(int)*1, hipMemcpyDeviceToHost));

	//Verify results
	if (gpu_OddCount == oc_check)
	{
		cout << "Number of Odds  determined by the GPU without Warp Primitives: " << gpu_OddCount << endl;
	}
	else {
		cout << "Number of Odds  determined by the GPU without Warp Primitives (Error):  " << gpu_OddCount << endl;
	}

	HandleCUDAError(hipFree(d_vect));
	HandleCUDAError(hipFree(d_oc));
	HandleCUDAError(hipDeviceReset());
}