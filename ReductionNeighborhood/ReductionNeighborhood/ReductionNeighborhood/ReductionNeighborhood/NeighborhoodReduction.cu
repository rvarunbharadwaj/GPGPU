#include "hip/hip_runtime.h"
﻿#include "ParallelReduction.h"
#include "GPUErrors.h"

__global__ void NeighborhoodWithDivergence(float* g_Vector, float* g_PartialSum)
{
	//Save threadIdx.x on the register
	int tid = threadIdx.x;

	//Compute the global thread index
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);

	//Compute the local pointer to each block
	float* blockAddress = g_Vector + (blockIdx.x * blockDim.x);

	//in-place reduction in global memory
	if (idx >= VECTOR_SIZE)
	{
		return;
	}
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();
	}
	if (tid == 0) //just threadIdx.x
	{
		g_PartialSum[blockIdx.x] = blockAddress[0]; //after for loop the final answer will be in 0 location
	}

}

__global__ void NeighborhoodWithLessDivergence(float* g_Vector, float* g_PartialSum)
{
	//Save threadIdx.x on the register
	int tid = threadIdx.x;

	//Compute the global thread index
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);

	//Compute the local pointer to each block
	float* blockAddress = g_Vector + (blockIdx.x * blockDim.x);

	//in-place reduction in global memory
	if (idx >= VECTOR_SIZE)
	{
		return;
	}
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		int index = 2 * stride * tid;
		if (index < blockDim.x)
		{
			blockAddress[index] += blockAddress[index + stride];
		}
		__syncthreads();
	}
	if (tid == 0) //just threadIdx.x
	{
		g_PartialSum[blockIdx.x] = blockAddress[0]; //after for loop the final answer will be in 0 location
	}

}

__host__ void OnNeighborhood(float* vectorTemp)
{
	chrono::time_point<std::chrono::system_clock> start, end;

	float* d_Vector;
	float* d_PartialSum;

	float* h_PartialSum;

	//Block and Thread Parameters
	dim3 block(256);
	dim3 grid((VECTOR_SIZE + block.x - 1) / block.x, 1);
	cout << "Neighborhood Implementations" << endl;
	cout << "\tThreads/Block: " << block.x << endl;
	cout << "\tBlocks/Grid: " << grid.x << endl;

	//The partial sums of each block
	h_PartialSum = new float[grid.x];

	//Allocate memory on the GPU to store the vector and partial sums
	HandleCUDAError(hipMalloc((void**)&d_Vector, VECTOR_SIZE_IN_BYTES));
	HandleCUDAError(hipMalloc((void**)&d_PartialSum, (grid.x * sizeof(float))));

	//Copy the vector to the GPU from the host
	HandleCUDAError(hipMemcpy(d_Vector, vectorTemp, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice));

	//Launch the Neighboorhood pairing kernel with Divergence
	NeighborhoodWithDivergence << <grid, block >> > (d_Vector, d_PartialSum);
	hipDeviceSynchronize();
	
	//Copy the vector to the GPU from the host containing the sum of each block
	HandleCUDAError(hipMemcpy(h_PartialSum, d_PartialSum, (grid.x * sizeof(float)), hipMemcpyDeviceToHost));
	//do reduction 
	float sum = 0.0f;
	for (int j = 0; j < grid.x; j++)
	{
		sum += h_PartialSum[j];
	}
	cout << "\t\tGPU Neighborhood Reduction: " << sum << endl;

	//Release Global Memory of d_Vector, and d_PartialSum
	HandleCUDAError(hipFree(d_Vector));
	HandleCUDAError(hipFree(d_PartialSum));

	//Reallocate Global Memory of d_Vector, and d_PartialSum
	HandleCUDAError(hipMalloc((void**)&d_Vector, VECTOR_SIZE_IN_BYTES));
	HandleCUDAError(hipMalloc((void**)&d_PartialSum, (grid.x * sizeof(float))));
	//Copy the vector to the GPU from the host
	HandleCUDAError(hipMemcpy(d_Vector, vectorTemp, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice));
	
	//Launch the Neighborhood pairing kernel with less divergence
	NeighborhoodWithLessDivergence << <grid, block >> > (d_Vector, d_PartialSum);
	hipDeviceSynchronize();

	//Copy the vector to the GPU from the host containing the sum of each block
	HandleCUDAError(hipMemcpy(h_PartialSum, d_PartialSum, (grid.x * sizeof(float)), hipMemcpyDeviceToHost));
	sum = 0.0f;
	for (int j = 0; j < grid.x; j++)
	{
		sum += h_PartialSum[j];
	}
	cout << "\t\tGPU Neighborhood Reduction: " << sum << endl;

	HandleCUDAError(hipFree(d_Vector));
	HandleCUDAError(hipFree(d_PartialSum));
	HandleCUDAError(hipDeviceReset());
}