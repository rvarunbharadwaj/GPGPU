#include "hip/hip_runtime.h"
#include "TransposeSMEM.h"
#include "GPUErrors.h"


__global__ void NaiveColTranspose(float* g_Matrix, float* g_MatrixTranspose, int ny, int nx)
{
	unsigned int ix = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned int iy = threadIdx.y + (blockIdx.y * blockDim.y);

	if (ix < nx && iy < ny)
	{
		g_MatrixTranspose[iy * nx + ix] = g_Matrix[ix * ny + iy];
	}
}

//Use of Shared Memory
#define ST_DIMX 16
#define ST_DIMY 16

__global__ void TransposeWithSM(float* g_Matrix, float* g_MatrixTranspose, int ny, int nx)
{
	//Declare static shared memory 
	__shared__ float tile[ST_DIMY][ST_DIMX];

	//Coordinates in original matrix
	unsigned int ix = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned int iy = threadIdx.y + (blockIdx.y * blockDim.y);

	//Linear global memory index address in the original matrix
	unsigned int ti = iy * nx + ix;

	//thread index in the transposed block
	unsigned int bidx = threadIdx.x + (threadIdx.y * blockDim.x);
	unsigned int irow = bidx / blockDim.y;
	unsigned int icol = bidx % blockDim.x;

	//Coordinates in transpose matrix
	ix = icol + (blockIdx.y * blockDim.y);
	iy = irow + (blockIdx.x * blockDim.x);

	//linear global memory index address in the transpose matrix
	unsigned int to = iy * ny + ix;

	if (ix < nx && iy < ny)
	{
		//Load the data from the original matrix into the tile on the shared memory
		tile[threadIdx.y][threadIdx.x] = g_Matrix[ti];
		__syncthreads();
		g_MatrixTranspose[to] = tile[icol][irow];
	}
}


__host__ void TransposeOnGPU(float* h_Matrix, float* h_MatrixTranspose, float* refTranspose, int ny, int nx)
{
	float* d_Matrix;
	float* d_MatrixTranspose;
	const int MatrixSizeInBytes = ny * nx * sizeof(float);

	//Allocate device memory on the global memory
	HandleCUDAError(hipMalloc((void**)&d_Matrix, MatrixSizeInBytes));
	HandleCUDAError(hipMalloc((void**)&d_MatrixTranspose, MatrixSizeInBytes));

	//transfer data from CPU Memory to GPU Memory
	HandleCUDAError(hipMemcpy(d_Matrix, h_Matrix, MatrixSizeInBytes, hipMemcpyHostToDevice));

	//Block and Grid Parameters
	int dimx = 16;
	int dimy = 16;
	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
	cout << "2D Grid Dimension" << endl;
	cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
	cout << "\tNumber of Blocks along Y dimension: " << grid.y << endl;
	cout << "2D Block Dimension" << endl;
	cout << "\tNumber of threads along X dimension: " << block.x << endl;
	cout << "\tNumber of threads along Y dimension: " << block.y << endl;

	//Matrix Transpose Load by Column (Stride Access) and Store by Row (Coalesced Access)
	NaiveColTranspose << <grid, block >> > (d_Matrix, d_MatrixTranspose, ny, nx);
	hipDeviceSynchronize();
	HandleCUDAError(hipMemcpy(h_MatrixTranspose, d_MatrixTranspose, MatrixSizeInBytes, hipMemcpyDeviceToHost));
	//Verify
	VerifyTranspose(h_MatrixTranspose, refTranspose, ny, nx);

	//Zero the computed transpose
	ZeroMatrix(h_MatrixTranspose, ny, nx);

	//Shared Memory based Transpose
	TransposeWithSM << <grid, block >> > (d_Matrix, d_MatrixTranspose, ny, nx);
	hipDeviceSynchronize();
	HandleCUDAError(hipMemcpy(h_MatrixTranspose, d_MatrixTranspose, MatrixSizeInBytes, hipMemcpyDeviceToHost));
	//Verify
	VerifyTranspose(h_MatrixTranspose, refTranspose, ny, nx);

	HandleCUDAError(hipFree(d_Matrix));
	HandleCUDAError(hipFree(d_MatrixTranspose));
	HandleCUDAError(hipDeviceReset());
}
