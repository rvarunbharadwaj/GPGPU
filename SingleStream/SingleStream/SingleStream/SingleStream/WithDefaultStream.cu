#include "hip/hip_runtime.h"
#include "VectorAddition.h"

__host__ void WithDefaultStream(float* h_A, float* h_B, float* h_C_GPU, const int nSize)
{
	float ElapsedTime{};
	//Create GPU Event Objects
	hipEvent_t start, stop;
	HandleCUDAError(hipEventCreate(&start));
	HandleCUDAError(hipEventCreate(&stop));

	float* dev_a{}, * dev_b{}, * dev_c{};
	//Allocate memory on the GPU using the hipMalloc function for the three vectors only for N elements
	HandleCUDAError(hipMalloc((void**)&dev_a, PARTITION_SIZE));
	HandleCUDAError(hipMalloc((void**)&dev_b, PARTITION_SIZE));
	HandleCUDAError(hipMalloc((void**)&dev_c, PARTITION_SIZE));

	//Creating number of threads/block and number of blocks
	int threads_PER_BLOCK = 256;
	int blocks_PER_GRID = (int)ceil(N / threads_PER_BLOCK);
	cout << "Vector Size = " << SIZE << endl;
	cout << "Number of Threads/Block for N = "<<N<<", " << threads_PER_BLOCK << endl;
	cout << "Number of Blocks/Grid for N = " << N << ", " << blocks_PER_GRID << endl;

	//Launch the kernel on the GPU to add the vectors in sections
	//Record the event time of the kernel execution starting
	HandleCUDAError(hipEventRecord(start, 0));
	for (unsigned int i = 0; i < SIZE; i += N)
	{
		//Copy only a section of the host data (h_A and h_B) to the device corresponding to size of N
		HandleCUDAError(hipMemcpy(dev_a, (h_A + i), PARTITION_SIZE, hipMemcpyHostToDevice));
		HandleCUDAError(hipMemcpy(dev_b, (h_B + i), PARTITION_SIZE, hipMemcpyHostToDevice));
		//Launch the kernel for adding only the section of size N
		AddVectors << <blocks_PER_GRID, threads_PER_BLOCK >> > (dev_a, dev_b, dev_c, N); //dev_a will be pointing to the first element
		
		hipDeviceSynchronize();
		//Copy only a section of the device data (dev_c) to the host corresponding to size of N
		HandleCUDAError(hipMemcpy((h_C_GPU + i), dev_c, PARTITION_SIZE, hipMemcpyDeviceToHost));
	}
	//Record the event time of the kernel execution completition
	HandleCUDAError(hipEventRecord(stop, 0));
	//Block the host to receive a synchronization recording event from the GPU
	HandleCUDAError(hipEventSynchronize(stop));

	//Compute the kernel execution time
	HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop));
	cout << "GPU Execution Non Stream Version: " << ElapsedTime << " msecs" << endl;

	//Release the allocated memory on the device
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	//Destroy the CUDA Event objects
	HandleCUDAError(hipEventDestroy(start));
	HandleCUDAError(hipEventDestroy(stop));

	//Reset the device before exiting for profiler tools like Nsight and Visual Profiler to show complete traces
	HandleCUDAError(hipDeviceReset());
}