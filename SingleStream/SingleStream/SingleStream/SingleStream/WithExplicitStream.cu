#include "hip/hip_runtime.h"
#include "VectorAddition.h"

__host__ void WithExplicitStream(const int nSize)
{
	float ElapsedTime{};
	//Determine whether the GPU Supports Streaming i.e., allows execution overlapping of kernels
	hipDeviceProp_t prop;
	int device_id;
	HandleCUDAError(hipGetDevice(&device_id));
	HandleCUDAError(hipGetDeviceProperties(&prop, device_id));
	cout << "Number of Asynchronous Engines: " << prop.asyncEngineCount << endl;
	if (!prop.concurrentKernels)
	{
		cout << "Device does not handle concurrent execution of kernels" << endl;
		return;
	}
	else {
		cout << "Device handles overlaps or streams" << endl; //if we copy from host to data on two streams both will be copied comcurrently. Only certain GPUs allow it.
	}

	float* h_A, * h_B, * h_C_CPU, * h_C_GPU;
	//Allocating pinned memory on the host using hipHostAlloc
	HandleCUDAError(hipHostAlloc((void**)&h_A, (SIZE * sizeof(float)), hipHostMallocDefault)); //will automatically allocate data with byte boundary
	HandleCUDAError(hipHostAlloc((void**)&h_B, (SIZE * sizeof(float)), hipHostMallocDefault));
	HandleCUDAError(hipHostAlloc((void**)&h_C_GPU, (SIZE * sizeof(float)), hipHostMallocDefault));
	HandleCUDAError(hipHostAlloc((void**)&h_C_CPU, (SIZE * sizeof(float)), hipHostMallocDefault));

	//Initialize Vectors
	InitializeVector(h_A, SIZE);
	InitializeVector(h_B, SIZE);
	CPUVectorAddition(h_A, h_B, h_C_CPU, SIZE); //size of h_A + h_B should be less than physical RAM

	//Device Allocations

	//Create GPU Event Objects
	hipEvent_t start, stop;
	HandleCUDAError(hipEventCreate(&start));
	HandleCUDAError(hipEventCreate(&stop));

	float* dev_a{}, * dev_b{}, * dev_c{};
	//Allocate memory on the GPU using the hipMalloc function for the three vectors only for N elements
	HandleCUDAError(hipMalloc((void**)&dev_a, PARTITION_SIZE));
	HandleCUDAError(hipMalloc((void**)&dev_b, PARTITION_SIZE));
	HandleCUDAError(hipMalloc((void**)&dev_c, PARTITION_SIZE));

	//Declaring a hipStream_t variable
	hipStream_t stream_1; //creating an object for explicit stream
	//Create an Explicit Stream Object
	HandleCUDAError(hipStreamCreate(&stream_1));

	//Creating number of threads/block and number of blocks
	int threads_PER_BLOCK = 256;
	int blocks_PER_GRID = (int)ceil(N / threads_PER_BLOCK);
	cout << "Vector Size = " << SIZE << endl;
	cout << "Number of Threads/Block for N = " << N << ", " << threads_PER_BLOCK << endl;
	cout << "Number of Blocks/Grid for N = " << N << ", " << blocks_PER_GRID << endl;

	//Launch the kernel on the GPU to add the vectors in sections using Stream and Asynchronously
	//Record the event time of the kernel execution starting
	HandleCUDAError(hipEventRecord(start, stream_1));
	for (unsigned int i = 0; i < SIZE; i += N)
	{
		//Copy Asynchronously only a section of the host data (h_A and h_B) to the device corresponding to size of N
		HandleCUDAError(hipMemcpyAsync(dev_a, (h_A + i), PARTITION_SIZE, hipMemcpyHostToDevice, stream_1)); //hipMemcpyAsync is a non blocking call
		HandleCUDAError(hipMemcpyAsync(dev_b, (h_B + i), PARTITION_SIZE, hipMemcpyHostToDevice, stream_1));
		//Launch the kernel on the explicit stream // third param with value 0 is shared memory size for dynamic alloc
		AddVectors << <blocks_PER_GRID, threads_PER_BLOCK, 0, stream_1 >> > (dev_a, dev_b, dev_c, N);
		//Copy Asynchronously only a section of the device data (dev_c) to the host corresponding to size of N
		HandleCUDAError(hipMemcpyAsync((h_C_GPU + i), dev_c, PARTITION_SIZE, hipMemcpyHostToDevice, stream_1));
	}
	//Block the host for GPU to synchronize with completition of the stream operations
	HandleCUDAError(hipStreamSynchronize(stream_1));

	//Record the event time of the kernel execution completition
	HandleCUDAError(hipEventRecord(stop, stream_1));
	//Block the host to receive a synchronization event of recording from the GPU
	HandleCUDAError(hipEventSynchronize(stop));

	HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop));
	cout << "GPU Execution Stream Version: " << ElapsedTime << " msecs" << endl;
	VerifyResults(h_C_CPU, h_C_GPU, SIZE);

	//Release the pinned memory on the host allocated
	HandleCUDAError(hipHostFree(h_A));
	HandleCUDAError(hipHostFree(h_B));
	HandleCUDAError(hipHostFree(h_C_CPU));
	HandleCUDAError(hipHostFree(h_C_GPU));

	//Destroy the stream object
	HandleCUDAError(hipStreamDestroy(stream_1));

	//Release the allocated memory on the device
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	//Destroy the event object
	HandleCUDAError(hipEventDestroy(start));
	HandleCUDAError(hipEventDestroy(stop));

	//Reset the device before exiting for profiler tools like Nsight and Visual Profiler to show complete traces
	HandleCUDAError(hipDeviceReset());
}