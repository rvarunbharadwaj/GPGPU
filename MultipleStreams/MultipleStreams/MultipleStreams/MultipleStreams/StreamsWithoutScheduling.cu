#include "hip/hip_runtime.h"
#include "VectorAddition.h"

__host__ void MultipleStreamsWOScheduling(const int nSize)
{
	//Allocating pinned memory on the host using hipHostAlloc
	float* h_A{}, * h_B{}, * h_C_CPU{}, * h_C_GPU{};
	HandleCUDAError(hipHostAlloc((void**)&h_A, (SIZE * sizeof(float)), hipHostMallocDefault));
	HandleCUDAError(hipHostAlloc((void**)&h_B, (SIZE * sizeof(float)), hipHostMallocDefault));
	HandleCUDAError(hipHostAlloc((void**)&h_C_CPU, (SIZE * sizeof(float)), hipHostMallocDefault));
	HandleCUDAError(hipHostAlloc((void**)&h_C_GPU, (SIZE * sizeof(float)), hipHostMallocDefault));

	//Initialize Vectors
	InitializeVector(h_A, SIZE);
	InitializeVector(h_B, SIZE);
	//Perform Addition on CPU using Pinned Memory
	CPUVectorAddition("Pinned",h_A, h_B, h_C_CPU, SIZE);

	//Device Allocations
	float ElapsedTime{};
	hipEvent_t start, stop;
	HandleCUDAError(hipEventCreate(&start));
	HandleCUDAError(hipEventCreate(&stop));

	//Declare a count variable of the device memory required
	unsigned int dev_count = 6;
	//Declare a vector of pointers
	float** dev{}; //Pointer to poniter
	//Allocate memory on the GPU using hipMalloc function for the required device memory 
	dev = new float* [dev_count];//Vector of pointers
	for (int i = 0; i < dev_count; i++)
	{
		HandleCUDAError(hipMalloc((void**)&dev[i], PARTITION_SIZE)); //Allocating 6 different memory segments of each PARTITION_SIZE
	}
	//Create an Explicit Multiple Streams
	hipStream_t stream_1, stream_2; //Create stream objects
	HandleCUDAError(hipStreamCreate(&stream_1));
	HandleCUDAError(hipStreamCreate(&stream_2));
	//Creating number of threads/block and number of blocks
	int threads_PER_BLOCK = 256;
	int blocks_PER_GRID = (int)ceil(N / threads_PER_BLOCK);
	cout << "Vector Size = " << SIZE << endl;
	cout << "Number of Threads/Block for N = " << N << ", " << threads_PER_BLOCK << endl;
	cout << "Number of Blocks/Grid for N = " << N << ", " << blocks_PER_GRID << endl;

	//Launch mutliple Streams to perform addition of the vectors overlapped without scheduling
	HandleCUDAError(hipEventRecord(start, 0));
	//Looping over full data using multiple streams
	for (unsigned int i = 0; i < SIZE; i += N*2)
	{
		//Copy the page locked or pinned memory of size N to the device memories for stream1 
		HandleCUDAError(hipMemcpyAsync(dev[0], (h_A + i), PARTITION_SIZE, hipMemcpyHostToDevice, stream_1));
		HandleCUDAError(hipMemcpyAsync(dev[1], (h_B + i), PARTITION_SIZE, hipMemcpyHostToDevice, stream_1));
		//Launch the kernel to perform addition on the size N of the vectors on stream 1
		AddVectors << <blocks_PER_GRID, threads_PER_BLOCK, 0, stream_1 >> > (dev[0], dev[1], dev[2], N);

		//Copy the the result device memory to the host pinned memory using stream1 
		HandleCUDAError(hipMemcpyAsync((h_C_GPU + i), dev[2], PARTITION_SIZE, hipMemcpyDeviceToHost, stream_1));

		//Copy the page locked or pinned memory of the next size N to the device memories using stream2 
		HandleCUDAError(hipMemcpyAsync(dev[3], (h_A + i + N), PARTITION_SIZE, hipMemcpyHostToDevice, stream_2));
		HandleCUDAError(hipMemcpyAsync(dev[4], (h_B + i + N), PARTITION_SIZE, hipMemcpyHostToDevice, stream_2));
		//Launch the kernel to perform addition on the next size N of the vectors on stream 2
		AddVectors << <blocks_PER_GRID, threads_PER_BLOCK, 0, stream_2 >> > (dev[3], dev[4], dev[5], N);
		//Copy the the result device memory to the host pinned memory using stream2 
		HandleCUDAError(hipMemcpyAsync((h_C_GPU + i + N), dev[5], PARTITION_SIZE, hipMemcpyDeviceToHost, stream_2));
	}
	HandleCUDAError(hipStreamSynchronize(stream_1));
	HandleCUDAError(hipStreamSynchronize(stream_2));
	HandleCUDAError(hipEventRecord(stop, 0));
	HandleCUDAError(hipEventSynchronize(stop));
	HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop));
	cout << "GPU Execution Stream Version: " << ElapsedTime << " msecs" << endl;
	VerifyResults(h_C_CPU, h_C_GPU, SIZE);

	//Release the pinned memory on the host allocated
	HandleCUDAError(hipHostFree(h_A));
	HandleCUDAError(hipHostFree(h_B));
	HandleCUDAError(hipHostFree(h_C_CPU));
	HandleCUDAError(hipHostFree(h_C_GPU));

	//Destroy the stream objects
	HandleCUDAError(hipStreamDestroy(stream_1));
	HandleCUDAError(hipStreamDestroy(stream_2));
	//Release the allocated memory on the device
	for (int i = 0; i < dev_count; i++)
	{
		HandleCUDAError(hipFree(dev[i]));
	}
	//Release the memory allocated on the host for the vector of pointers
	delete[] dev;

	//Destroy the event object
	HandleCUDAError(hipEventDestroy(start));
	HandleCUDAError(hipEventDestroy(stop));

	//Reset the device before exiting for profiler tools like Nsight and Visual Profiler to show complete traces
	HandleCUDAError(hipDeviceReset());
}