#include "hip/hip_runtime.h"
#include "RGBtoGrayScale.h"
//Kernel Version 0
__global__ void gpu_RGBtoGrayScaleVer0(unsigned char* in, unsigned char* out, unsigned int h, unsigned int w)
{
	unsigned int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	if (idx < (w * h))
	{
		out[idx] = 0.21f * in[idx] + 0.71f * in[idx + h * w] + 0.07f * in[idx + 2 * h * w];
	}
}
//Kernel Version 1
__global__ void gpu_RGBtoGrayScaleVer1(unsigned char* r,
	unsigned char* g,
	unsigned char* b,
	unsigned char* out, unsigned int h, unsigned int w)
{
	unsigned int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	if (idx < (w * h))
	{
		out[idx] = 0.21f * *(r + idx) + 0.72f * *(g + idx) + 0.07f * *(b + idx);
	}
}
//Kernel Version 2: 2D implementation
__global__ void gpu_RGBtoGrayScaleVer2(unsigned char* r,
	unsigned char* g,
	unsigned char* b,
	unsigned char* out, unsigned int h, unsigned int w)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	//Compute the grayscale image index 1) y*w points to beginning of each row
	int idx = y * w + x;
	if (x < w && y < h)
	{
		out[idx] = 0.21f * *(r + idx) + 0.72f * *(g + idx) + 0.07f * *(b + idx);
	}
}

//Host Helper function
__host__ double gpu_RGBtoGrayScaleHelper(unsigned char* h_in, unsigned char* h_out, 
	unsigned int rgbSIZE,
	unsigned int graySIZE,
	unsigned int h,
	unsigned int w,
	unsigned int kernelVer)
{
	double computeTime;
	unsigned char* d_in, * d_out;
	//Allocating device memory for the RGB and GrayScale Images

	if (!HandleCUDAError(hipMalloc((void**) & d_in, rgbSIZE)))
	{
	cout << "Error allocating memory on the gpu for the rgb image" << endl;
	return FALSE;
	}

	if (!HandleCUDAError(hipMalloc((void**) & d_out, rgbSIZE)))
	{
	cout << "Error allocating memory on the gpu for the gray image" << endl;
	return FALSE;
	}
	//Copying the RGB image to the device
	if (!HandleCUDAError(hipMemcpy(d_in,h_in,rgbSIZE,hipMemcpyHostToDevice)))
	{
	cout << "Error trasferring image on to gpu" << endl;
	return FALSE;
	}
	//Setup Execution Configuration Parameters
	unsigned int threadsPerBlock = 256;
	unsigned int blocksPerGrid = ((w * h)/threadsPerBlock)+1;
	
	cout << "Image Grid Size = " << (w * h) << " pixels" << endl;
	cout << "Number of threads per block = " << threadsPerBlock << endl;
	cout << "Number of blocks per Grid = " << blocksPerGrid << endl;
	cout << "Total Number of Threads in the Grid = " << threadsPerBlock * blocksPerGrid << endl;
	
	if (kernelVer == 0)
	{
		//Launch the RGB to Gray Scale Kernel - Ver 0
		auto start = high_resolution_clock::now();
		gpu_RGBtoGrayScaleVer0 << <blocksPerGrid, threadsPerBlock >> > (d_in,
			d_out,
			h,
			w);
		hipDeviceSynchronize();
		auto end = high_resolution_clock::now();
		auto elasped_seconds = end - start;
		computeTime = duration_cast<microseconds>(elasped_seconds).count();
	}
	else if(kernelVer == 1)
	{
		unsigned char* d_r = d_in;
		unsigned char* d_g = d_in + h * w;
		unsigned char* d_b= d_in + 2 * h * w;
		//Launch the RGB to Gray Scale Kernel - Ver 1
		auto start = high_resolution_clock::now();
		gpu_RGBtoGrayScaleVer1 << <blocksPerGrid, threadsPerBlock >> > (d_r,
			d_g,
			d_b,
			d_out,
			h,
			w);
		hipDeviceSynchronize();
		auto end = high_resolution_clock::now();
		auto elasped_seconds = end - start;
		computeTime = duration_cast<microseconds>(elasped_seconds).count();
	}
	else 
	{
		//2D Version
		unsigned char* d_r = d_in;
		unsigned char* d_g = d_in + h * w;
		unsigned char* d_b = d_in + 2 * h * w;
		//Setup Execution Configuration Parameters
		int TILE_WIDTH = 16;
		dim3 dimGrid(ceil((float)w / TILE_WIDTH), ceil((float)h / TILE_WIDTH));
		dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
		auto start = high_resolution_clock::now();
		gpu_RGBtoGrayScaleVer2 << <dimGrid, dimBlock >> > (d_r,
			d_g,
			d_b,
			d_out,
			h,
			w);
		hipDeviceSynchronize();
		auto end = high_resolution_clock::now();
		auto elasped_seconds = end - start;
		computeTime = duration_cast<microseconds>(elasped_seconds).count();
	}
	//Copy the grayscale image data from device to host
	if (!HandleCUDAError(hipMemcpy(h_out, d_out, graySIZE, hipMemcpyDeviceToHost)))
	{
		cout << "Error trasferring image on to gpu" << endl;
		return FALSE;
	}
	
	if (!HandleCUDAError(hipFree(d_in)))
	{
		cout << "Error freeing RGB image memory" << endl;
		return FALSE;
	}
	if (!HandleCUDAError(hipFree(d_out)))
	{
		cout << "Error freeing GrayScale image memory" << endl;
		return FALSE;
	}
	HandleCUDAError(hipDeviceReset());
	return computeTime;
}